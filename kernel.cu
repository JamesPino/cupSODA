#include "hip/hip_runtime.h"
/*

kernel.cu: cupSODA's main file.

cupSODA is black-box deterministic simulator of biological systems that 
exploits the remarkable memory bandwidth and computational capability of GPUs.
cupSODA allows to efficiently execute in parallel large numbers of simulations, 
which are usually required to investigate the emergent dynamics of a given 
biological system under different conditions. cupSODA works by automatically 
deriving the system of ordinary differential equations from a reaction-based 
mechanistic model, defined according to the mass-action kinetics, and then 
exploiting the numerical integration algorithm, LSODA.

See file COPYING for copyright and licensing information.

Bibliography:

- Nobile M.S., Cazzaniga P., Besozzi D., Mauri G.: GPU-accelerated simulations of
mass-action kinetics models with cupSODA. The Journal of Supercomputing, 
vol. 69, issue 1, pp.17�24, 2014

- Petzold L.: Automatic selection of methods for solving stiff and nonstiff 
systems of ordinary differential equations. SIAM Journal of Scientific and
Statistical Computing, 4(1):136�148, 1983

Direct link to the cupSODA paper: http://link.springer.com/article/10.1007/s11227-014-1208-8

*/

#include <stdio.h>
#include <math.h>
#include <iostream>
#include <fstream>
#include <vector>
#include "constants.h"
#include "service_stuff.h"
#include "cupSODA.h"
#include "input_reader.h"
#include "stoc2det.h"
#include <hip/hip_runtime.h>


// #define VERBOSE
#define DUMP

#define USE_NEW_ODE


/* Pointer to compressed ODE on the GPU */
char* device_compressed_odes;

/* Int2String conversion utility */
std::string convertInt(int number)
{
   std::stringstream ss;//create a stringstream
   ss << number;//add number to the stream
   return ss.str();//return a string with the contents of the stream
}

int main(int argc, char** argv)  
{

	/* This function verifies that all non-optional arguments are correctly passed to cupSODA */
	if ( !CheckArguments(argc, argv)) {
		exit(-1);
	}
	
	/* The following code determines and sets the optional values. */
	unsigned int GPU = atoi(argv[5]);

	bool just_fitness = true;
	bool print_fitness = false;

	if (argc>6) just_fitness  = std::string(argv[6])!="0";
	if (argc>6) print_fitness = std::string(argv[6])=="2";
	
	bool ACTIVATE_SHARED_MEMORY = false;
	bool ACTIVATE_CONSTANT_MEMORY = false;
	
	if (argc>7) {
		ACTIVATE_SHARED_MEMORY = std::string(argv[7])!="0";
		ACTIVATE_CONSTANT_MEMORY = std::string(argv[7])=="2";
	}

	bool DUMP_DEBUG = false;

	if (argc>8) {
		DUMP_DEBUG = std::string(argv[8])!="0";
	}



	/* The st2det object load and contains all the information related to the model. */
	st2det* s2d = new st2det();
	s2d->blocks = atoi(argv[2]);		// TODO
	bool a = s2d->open_files(argv[1], argv[3], argv[4], true, DUMP_DEBUG, just_fitness, ACTIVATE_CONSTANT_MEMORY);
	
	if (!print_fitness) {
		if ( just_fitness ) {
			printf(" * Fitness calculation: ENABLED \n");	
		} else {
			printf(" * Fitness calculation: DISABLED \n");
		}
	}
	
	if (a!=true) {
		perror("ERROR while opening input files:");
		exit(-3);
	} else {
		if (!print_fitness) 		printf(" * All files correct\n");		
	}

	/* This function sets all the constant values in the GPU, storing them in constant memory. */
	SetConstants(s2d->species, s2d->reactions, s2d->ODE_lun, s2d->JAC_lun, s2d->species_to_sample.size(), s2d->time_instants.size(), s2d->repetitions, s2d->experiments, s2d->threads, false);

	param_t* device_flattenODE;
	unsigned int* device_offsetODE;
	param_t* device_flattenJAC;
	unsigned int* device_offsetJAC;
	unsigned int* device_species_to_sample;

	conc_t* device_X;
	conc_t* host_X;
	conc_t* device_target;

	/* Allocate and store the ODE system and the Jacobian matrix */
	hipMalloc((void**)&device_flattenODE,sizeof(param_t)*s2d->ODE_lun);
	CudaCheckError();
	hipMemcpy(device_flattenODE,s2d->ODE,sizeof(param_t)*s2d->ODE_lun,hipMemcpyHostToDevice);
	CudaCheckError();
	hipMalloc((void**)&device_flattenJAC,sizeof(param_t)*s2d->JAC_lun);
	CudaCheckError();
	hipMemcpy(device_flattenJAC,s2d->JAC,sizeof(param_t)*s2d->JAC_lun,hipMemcpyHostToDevice);
	CudaCheckError();


	// Allocate and store the offsets used by the ODE and Jacobian representations */
	hipMalloc((void**)&device_offsetODE,sizeof(unsigned int)*s2d->species);
	CudaCheckError();
	hipMemcpy(device_offsetODE,s2d->ODE_offset,sizeof(unsigned int)*s2d->species,hipMemcpyHostToDevice);
	CudaCheckError();	
	hipMalloc((void**)&device_offsetJAC,sizeof(unsigned int)*s2d->species*s2d->species);
	CudaCheckError();
	hipMemcpy(device_offsetJAC,s2d->JAC_offset,sizeof(unsigned int)*s2d->species*s2d->species,hipMemcpyHostToDevice);
	CudaCheckError();


	/* Allocates the memory space for samples and the target time-series */
	hipMalloc((void**)&device_X, sizeof( conc_t ) * s2d->species_to_sample.size() * s2d->time_instants.size() * s2d->threads );
	CudaCheckError();
	hipMalloc((void**)&device_target, sizeof( conc_t ) * s2d->repetitions * s2d->experiments * s2d->target_quantities * s2d->time_instants.size() );
	CudaCheckError();
	hipMemcpy(device_target,s2d->global_time_series, sizeof( conc_t ) * s2d->repetitions * s2d->experiments * s2d->target_quantities * s2d->time_instants.size() ,hipMemcpyHostToDevice);
	CudaCheckError();
	host_X = (conc_t*) malloc ( sizeof( conc_t ) * s2d->species_to_sample.size() * s2d->time_instants.size() * s2d->threads );
	if (host_X==NULL) {
		perror("ERROR allocating states\n");
		exit(-14);
	}
	memset( host_X, 0, sizeof( conc_t ) * s2d->species_to_sample.size() * s2d->time_instants.size() * s2d->threads );
	
	/* Load the initial state of the system, according to input files */
	for (unsigned int ss=0; ss<s2d->species_to_sample.size(); ss++) {
		for (unsigned int t=0; t<s2d->threads; t++) {
			host_X[ s2d->threads*ss + t ] = s2d->X[ s2d->species_to_sample[ss] ]; 
		}
	}	
	hipMemcpy(device_X,host_X,sizeof(conc_t) * s2d->species_to_sample.size() * s2d->time_instants.size() * s2d->threads, hipMemcpyHostToDevice );
	CudaCheckError();
	

	/* Allocate and store the species to be sampled */
	hipMalloc((void**)&device_species_to_sample, sizeof( unsigned int ) * s2d->species_to_sample.size() );
	hipMemcpy(device_species_to_sample,&s2d->species_to_sample[0],sizeof(unsigned int) * s2d->species_to_sample.size(),hipMemcpyHostToDevice);
	CudaCheckError();
	
		
	///// DEBUG //////
	int* h_debug = (int*) malloc ( sizeof(int)*s2d->threads );
	int* d_debug;
	hipMalloc((void**)&d_debug,sizeof(int)*s2d->threads);	
	CudaCheckError();  
	///// DEBUG //////

	
	 /* Local variables */
	 double* constants = (double*) malloc( sizeof(double) *s2d->threads*s2d->reactions );
     double *t = (double*)malloc(sizeof(double)*s2d->threads);
	 double *y = (double*)malloc(sizeof(double)*s2d->species*s2d->threads);
     int *jt = (int*)malloc(sizeof(int)*s2d->threads);
     int *neq = (int*)malloc(sizeof(int)*s2d->threads);
	 int *liw = (int*)malloc(sizeof(int)*s2d->threads);
	 int *lrw = (int*)malloc(sizeof(int)*s2d->threads);     
	 double *atol = (double*)malloc(sizeof(double)*s2d->species*s2d->threads);
     int *itol =(int*) malloc(sizeof(int)*s2d->threads);
	 int *iopt =(int*) malloc(sizeof(int)*s2d->threads);
     double *rtol = (double*)malloc(sizeof(double)*s2d->threads);
     int *iout =(int*) malloc(sizeof(int)*s2d->threads);
     double *tout =(double*) malloc(sizeof(double)*s2d->threads);
     int *itask = (int*)malloc(sizeof(int)*s2d->threads);
		 	 
	 // const int lrn = 20 + 16*s2d->species;
	 const int lrs = 22+9*s2d->species + (s2d->species*s2d->species);
	 // const int LRW = max(lrn,lrs);
	 const int LRW = 22+s2d->species*max(16,s2d->species+9);	 
	 const int LIW = 20+s2d->species;	 
	 
	 
	double *rwork = (double*)malloc(sizeof(double)*LRW*s2d->threads);
	memset(rwork, 0, sizeof(double)*LRW*s2d->threads); // TEST

	int *iwork = (int*) malloc(sizeof(int)*LIW*s2d->threads);     
	memset(iwork, 0, sizeof(int)*LIW*s2d->threads); // TEST

	 int *istate = (int*)malloc(sizeof(int)*s2d->threads);
	struct cuLsodaCommonBlock* common = (struct cuLsodaCommonBlock*) malloc(sizeof(struct cuLsodaCommonBlock)*s2d->threads);
	struct cuLsodaCommonBlock *Hcommon = common;


	/* Pointers to GPU's global memory areas of LSODA data structures */
	double* device_constants;
	double	*_Dt;
	double	*_Dy;	
	int	*_Djt;
	int	*_Dneq;
	int	*_Dliw;
	int	*_Dlrw;
    double	*_Datol;
    int	*_Ditol;
	int	*_Diopt;
    double	*_Drtol;
    int	*_Diout;
    double	*_Dtout;
    int	*_Ditask;
	int	*_Diwork;	
    double	*_Drwork;	
	int	*_Distate;
	struct cuLsodaCommonBlock *_Dcommon;	
	
	/* Method instantiations for Derivative and Jacobian functions to send to template */
	myFex fex;
	myJex jex;

	/* Assignment of initial values to locals */
	for (unsigned int i=0; i<s2d->threads; i++) {
		
		 // iwork[i*LIW+5] = 100000;		// default
		iwork[i*LIW+5] = s2d->max_steps;

		for (unsigned r=0; r<s2d->reactions; r++) 
			constants[i*s2d->reactions+r] = s2d->c_matrix[ i*s2d->reactions+r ];

		/* Number of ODEs */
		neq[i] = s2d->species;

		/* Initial quantities */
		for (unsigned int s=0; s<s2d->species; s++)
			y[i*s2d->species+s] = s2d->X[i*s2d->species+s];

		/* Initial time */
		t[i] = (double)0.;
				
		/* Error tolerances */
		itol[i] = 2;
		rtol[i] = s2d->rtol;						
		for (unsigned int s=0; s<s2d->species; s++) {
			atol[i*s2d->species+s] = s2d->atol[s];			
		}		

		/* Standard LSODA execution */
		itask[i] = 1;
		istate[i] = 1;
		iopt[i] = 0;
		lrw[i] = LRW;
		liw[i] = LIW;
		jt[i] = 2;
	}
	cuLsodaCommonBlockInit(Hcommon, s2d->threads);
	
	/* Allocate the global memory for LSODA data structures, replicated for each thread,
	   and store the values loaded from local files. */
	hipMalloc((void**)&device_constants,sizeof(double)*s2d->threads*s2d->reactions);	hipMemcpy(device_constants,constants,sizeof(double)*s2d->threads*s2d->reactions,hipMemcpyHostToDevice);
	hipMalloc((void**)&_Dt,sizeof(double)*s2d->threads);								hipMemcpy(_Dt,t,sizeof(double)*s2d->threads,hipMemcpyHostToDevice);
	hipMalloc((void**)&_Dy,sizeof(double)*s2d->species*s2d->threads);					hipMemcpy(_Dy,y,sizeof(double)*s2d->species*s2d->threads,hipMemcpyHostToDevice);
	hipMalloc((void**)&_Djt,sizeof(int)*s2d->threads);									hipMemcpy(_Djt,jt,sizeof(int)*s2d->threads,hipMemcpyHostToDevice);
	hipMalloc((void**)&_Dneq,sizeof(int)*s2d->threads);								hipMemcpy(_Dneq,neq,sizeof(int)*s2d->threads,hipMemcpyHostToDevice);
	hipMalloc((void**)&_Dliw,sizeof(int)*s2d->threads);								hipMemcpy(_Dliw,liw,sizeof(int)*s2d->threads,hipMemcpyHostToDevice);
	hipMalloc((void**)&_Dlrw,sizeof(int)*s2d->threads);								hipMemcpy(_Dlrw,lrw,sizeof(int)*s2d->threads,hipMemcpyHostToDevice);
	hipMalloc((void**)&_Datol,sizeof(double)*s2d->species*s2d->threads);				hipMemcpy(_Datol,atol,sizeof(double)*s2d->species*s2d->threads,hipMemcpyHostToDevice);
	hipMalloc((void**)&_Ditol,sizeof(int)*s2d->threads);								hipMemcpy(_Ditol,itol,sizeof(int)*s2d->threads,hipMemcpyHostToDevice);
	hipMalloc((void**)&_Diopt,sizeof(int)*s2d->threads);								hipMemcpy(_Diopt,iopt,sizeof(int)*s2d->threads,hipMemcpyHostToDevice);
	hipMalloc((void**)&_Drtol,sizeof(double)*s2d->threads);							hipMemcpy(_Drtol,rtol,sizeof(double)*s2d->threads,hipMemcpyHostToDevice);
	hipMalloc((void**)&_Diout,sizeof(int)*s2d->threads);								hipMemcpy(_Diout,iout,sizeof(int)*s2d->threads,hipMemcpyHostToDevice);
	hipMalloc((void**)&_Dtout,sizeof(double)*s2d->threads);							hipMemcpy(_Dtout,tout,sizeof(double)*s2d->threads,hipMemcpyHostToDevice);
	hipMalloc((void**)&_Ditask,sizeof(int)*s2d->threads);								hipMemcpy(_Ditask,itask,sizeof(int)*s2d->threads,hipMemcpyHostToDevice);
	hipMalloc((void**)&_Diwork,sizeof(int)*LIW*s2d->threads);							hipMemcpy(_Diwork,iwork,sizeof(int)*LIW*s2d->threads,hipMemcpyHostToDevice);
	hipMalloc((void**)&_Drwork,sizeof(double)*LRW*s2d->threads);						hipMemcpy(_Drwork,rwork,sizeof(double)*LRW*s2d->threads,hipMemcpyHostToDevice);
	hipMalloc((void**)&_Distate,sizeof(int)*s2d->threads);								hipMemcpy(_Distate,istate,sizeof(int)*s2d->threads,hipMemcpyHostToDevice);
	hipMalloc((void**)&_Dcommon,sizeof(struct cuLsodaCommonBlock)*s2d->threads);		hipMemcpy(_Dcommon,Hcommon,sizeof(struct cuLsodaCommonBlock)*s2d->threads, hipMemcpyHostToDevice);
	CudaCheckError()  ;
	
	unsigned int sh_memory_bytes;

	/* Check for available shared memory: if the execution hierarchy (i.e., number of threads per block)
	   cannot be launched with the current configuration, then abort. */
	if (ACTIVATE_SHARED_MEMORY) {
		sh_memory_bytes = sizeof(double)*s2d->species*s2d->tpb + sizeof(double)*s2d->tpb;
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, GPU);
		if (sh_memory_bytes > prop.sharedMemPerBlock ) {
			printf("ERROR: insufficient shared memory (%d).\n", sh_memory_bytes);
			exit(ERROR_INSUFF_SHARED_MEMORY);
		}
	} else {
		sh_memory_bytes = 0;
	}

	SetODEarray( s2d );

	// Code for profiling
	hipEvent_t start, stop;
	if (!just_fitness) {
		hipEventCreate(&start);  
		hipEventCreate(&stop);
		hipEventRecord(start, 0);
	}
	
	// LSODA documentation reads: "write a main program which calls subroutine lsoda once for each point at which answers are desired"
	// For this reason, I use a for cycle that goes through the set of sampling temporal instants.
	for (unsigned int ti=0; ti<s2d->time_instants.size(); ti++) 
	{

		/* Si pu� parallelizzare: TODO */		
		for (unsigned int i=0; i<s2d->threads; i++) {
			tout[i] = s2d->time_instants[ti];
		}	
		hipMemcpy(_Dtout,tout,sizeof(double)*s2d->threads,hipMemcpyHostToDevice);
					
		dim3 BlocksPerGrid(s2d->blocks,1,1);
		dim3 ThreadsPerBlock(s2d->tpb,1,1);
		
		// CUDA LSODA entry point 
		cuLsoda<<<BlocksPerGrid,ThreadsPerBlock,sh_memory_bytes>>>
			(fex, _Dneq, _Dy, _Dt, _Dtout, _Ditol, _Drtol, _Datol, _Ditask, _Distate, _Diopt, _Drwork, _Dlrw, _Diwork, _Dliw, 
			 jex, _Djt, _Dcommon, d_debug, device_compressed_odes, device_flattenODE, device_offsetODE, device_constants, device_X, ti, 
			 device_species_to_sample, device_flattenJAC, device_offsetJAC, ACTIVATE_SHARED_MEMORY, ACTIVATE_CONSTANT_MEMORY);
		CudaCheckError();

		/* Print debug information (if requested), for each thread */
		if (DUMP_DEBUG) {
			hipMemcpy(istate,_Distate, sizeof(int)*s2d->threads,hipMemcpyDeviceToHost);
			printf(" * Dumping istates:\n");
			for (unsigned int th=0; th<s2d->threads; th++) {				
				printf("Thread %d: istate %d", th, istate[th]);
				switch(istate[th]) {
				case 1: printf(" (First step) "); break; 
				case 2: printf(" (OK!) "); break; 
				case -1: printf (" (excess of word done) "); break;
				case -2: printf (" (excess of accuracy requested) "); break;
				case -3: printf (" (illegal input detected) "); break;
				case -4: printf (" (repeated error test failures) "); break;
				case -5: printf (" (convergence failure) "); break;
				case -6: printf (" (error weight became zero) "); break;
				case -7: printf (" (work space insufficient to finish) "); break;
				default:
					printf(" (UNKNOWN LSODA ERROR) "); break;
				};
				printf("\n");
			}

			printf("\n");
		}

    }

	if (!just_fitness) {
		hipEventRecord( stop, 0 );
		hipEventSynchronize( stop );
		float tempo;
		hipEventElapsedTime( &tempo, start, stop );
		tempo /= 1000;
		printf("Running time: %f seconds\n", tempo);
		hipEventDestroy(start); 
		hipEventDestroy(stop); 
	}
	
	unsigned int DEV_CONST_SAMPLESLUN = s2d->species_to_sample.size();

	hipDeviceSynchronize();

	/* If we are just calculating a fitness value, avoid creating and dumping to output files of simulations */
	if (just_fitness) {

		double* device_fitness;
		double* host_fitness;
		char* device_swarms;

		host_fitness = (double*) malloc ( sizeof(double) * s2d->threads );

		hipMalloc((void**)&device_fitness,sizeof(double)*s2d->threads);	
		hipMalloc((void**)&device_swarms, sizeof(char)  *s2d->threads);	
		CudaCheckError();

		hipMemcpy(device_swarms, s2d->thread2experiment, sizeof(char)*s2d->threads, hipMemcpyHostToDevice);
		CudaCheckError();
		
		dim3 BlocksPerGrid(s2d->blocks,1,1);
		dim3 ThreadsPerBlock(s2d->tpb,1,1);

		calculateFitness<<<BlocksPerGrid,ThreadsPerBlock>>>( device_X, device_target, device_fitness, device_swarms );
		CudaCheckError();

		// hipMemcpy(device_swarms,s2d->thread2experiment,sizeof(char)*s2d->threads, hipMemcpyDeviceToHost);
		hipMemcpy(host_fitness,device_fitness,sizeof(double)*s2d->threads, hipMemcpyDeviceToHost);
		CudaCheckError();

		// if we are not just printing on video the fitness values, open the output file "pref_allfit"
		if ( !print_fitness ) {

		#ifdef _WIN32
			std::string comando_crea_cartella("md ");
			comando_crea_cartella.append(s2d->DEFAULT_OUTPUT);		
			system(comando_crea_cartella.c_str());
		#else
			std::string comando_crea_cartella("mkdir ");
			comando_crea_cartella.append(s2d->DEFAULT_OUTPUT);
			system(comando_crea_cartella.c_str());
		#endif


			std::string outputfile(s2d->DEFAULT_OUTPUT);
			outputfile.append("/pref_allfit");
		
			std::ofstream dump2(outputfile.c_str());

			if (!dump2.is_open()) {
				printf("Path: %s.\n", outputfile.c_str());
				perror("Unable to save fitness file 'pref_allfit', aborting.");
				exit(-17);
			}

			// verify!!
			for (unsigned int sw=0; sw<s2d->blocks; sw++) {
				for (unsigned int part=0; part<s2d->tpb; part++) {
					dump2 << host_fitness[sw*s2d->tpb + part] << "\t";					
				}
				dump2 << "\n";
			}
			dump2.close();

		} else {

			// verify!!
			for (unsigned int sw=0; sw<s2d->blocks; sw++) {
				for (unsigned int part=0; part<s2d->tpb; part++) {
					std::cout << host_fitness[sw*s2d->tpb + part] << "\t";					
				}
				std::cout << "\n";
			}

		}
		

		// libero memoria
		free(host_X);
		free(host_fitness);
	
		// libero memoria in GPU (TODO)
		hipFree(device_X);
		hipFree(device_compressed_odes);
		hipFree(device_constants);
		hipFree(device_flattenJAC);
		hipFree(device_flattenODE);
		hipFree(device_offsetJAC);
		hipFree(device_offsetODE);
		hipFree(device_species_to_sample);	
		hipFree(device_fitness);	
		hipFree(device_swarms);
		hipFree(device_target);

		exit(0);
	} 


	// No fitness: let's output the simulations
	hipMemcpy(host_X,device_X, sizeof(conc_t) * s2d->species_to_sample.size() * s2d->threads * s2d->time_instants.size(), hipMemcpyDeviceToHost);

	#ifdef _WIN32
		std::string comando_crea_cartella("md ");
		comando_crea_cartella.append(s2d->DEFAULT_OUTPUT);		
		system(comando_crea_cartella.c_str());
	#else
		std::string comando_crea_cartella("mkdir ");
		comando_crea_cartella.append(s2d->DEFAULT_OUTPUT);
		system(comando_crea_cartella.c_str());
	#endif

	for ( unsigned int tid=0; tid<s2d->actual_threads; tid++ ) {
				
		std::string outputfile(s2d->DEFAULT_OUTPUT);
		outputfile.append("/");
		outputfile.append(s2d->DEFAULT_PREFIX);
		outputfile.append("_");
		outputfile.append( convertInt(tid) );

		std::ofstream dump2(outputfile.c_str());

		if (! dump2.good() ) {
			perror("ERROR: cannot save dynamics");
			// system("pause");
			exit(-2);
		}
	 
		
		unsigned int larg = s2d->threads;
		unsigned int DEV_CONST_SAMPLESLUN = s2d->species_to_sample.size();
		
	
		for (unsigned int campione=0; campione<s2d->time_instants.size(); campione++) {

			dump2 << s2d->time_instants[campione] << "\t";

			for (unsigned int s=0; s<s2d->species_to_sample.size(); s++) {

				// ACCESS_SAMPLE = larg*DEV_CONST_SAMPLESLUN*campione + larg*s + tid 
				
				dump2 << host_X[ ACCESS_SAMPLE ];
				if ( s!=s2d->species_to_sample.size()-1 )
					dump2 << "\t";

			}

			dump2 << "\n";

		}

		// dump2 << "\n\n";


		dump2.close();

	}

	// Free local memory
	free(host_X);
	
	// Free GPU's memory
	hipFree(device_X);
	hipFree(device_compressed_odes);
	hipFree(device_constants);
	hipFree(device_flattenJAC);
	hipFree(device_flattenODE);
	hipFree(device_offsetJAC);
	hipFree(device_offsetODE);
	hipFree(device_species_to_sample);	

    return 0;
} 


